#include "hip/hip_runtime.h"
#include "cme_util.h"

namespace cuFSP{

    __device__ __host__
    void indx2state(size_t indx, int *state, size_t dim, size_t *fsp_bounds) {
        for (size_t i{1}; i <= dim; i++) {
            state[i - 1] = indx % (fsp_bounds[i - 1] + 1);
            indx = indx / (fsp_bounds[i - 1] + 1);
        }
    }

    __device__ __host__
    int state2indx(int *state, size_t dim, size_t *fsp_bounds) {
        int indx = 0;
        int nprod = 1;
        for (size_t i{1}; i <= dim; ++i) {
            indx += state[i - 1] * nprod;
            nprod *= (fsp_bounds[i - 1] + 1);
        }
        return indx;
    }

    __global__
    void fsp_get_states(int *d_states, size_t dim, size_t n_states, size_t *n_bounds) {

        extern __shared__
        size_t n_bounds_copy[];

        size_t ti = threadIdx.x;
        size_t indx = blockIdx.x * blockDim.x + ti;

        if (ti < dim) {
            n_bounds_copy[ti] = n_bounds[ti];
        }

        __syncthreads();

        if (indx < n_states) {
            indx2state(indx, &d_states[indx * dim], dim, &n_bounds[0]);
        }
    }

    __host__
    __device__
    void reachable_state(int *state, int *rstate, int reaction, int direction,
                         int n_species, int *stoich_val, int *stoich_colidxs, int *stoich_rowptrs) {
        for (int k{0}; k < n_species; ++k) {
            rstate[k] = state[k];
        }
        for (int i = stoich_rowptrs[reaction]; i < stoich_rowptrs[reaction + 1]; ++i) {
            rstate[stoich_colidxs[i]] += direction * stoich_val[i];
        }
    }
}
