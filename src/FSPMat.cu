#include "hip/hip_runtime.h"
#include "FSPMat.h"
#include "../../../../../../usr/local/cuda/include/hip/driver_types.h"

using namespace arma;

namespace cuFSP {

    size_t FSPMat::get_n_rows() {
        return nst;
    }

    size_t FSPMat::get_n_species() {
        return ns;
    }

    size_t FSPMat::get_n_reactions() {
        return nr;
    }

    cuda_csr_mat *FSPMat::get_term(size_t i) {
        return &term[i];
    }

// Constructor
    // Precondition:
    // stoich stores the stoichiometry matrix, assumed to be in CSR format, with each row for each reaction
    FSPMat::FSPMat
//    (hipsparseHandle_t _handle, hipStream_t _stream,
            (int *states, size_t n_states, size_t n_reactions, size_t n_species, size_t *fsp_dim,
                   cuda_csr_mat_int stoich, TcoefFun t_func, PropFun prop_func) {

//        cusparse_handle = _handle;
//        stream = _stream;

        std::cout << "n_states = " << n_states << "\n";

        int *d_stoich_vals, *d_stoich_colidxs, *d_stoich_rowptrs;
        hipMalloc(&d_stoich_vals, stoich.row_ptrs[stoich.n_rows] * sizeof(int));
        hipMalloc(&d_stoich_colidxs, stoich.row_ptrs[stoich.n_rows] * sizeof(int));
        hipMalloc(&d_stoich_rowptrs, (stoich.n_rows + 1) * sizeof(int));

        hipMemcpy(d_stoich_vals, stoich.vals, stoich.row_ptrs[stoich.n_rows] * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_stoich_colidxs, stoich.col_idxs, stoich.row_ptrs[stoich.n_rows] * sizeof(int),
                   hipMemcpyHostToDevice);
        hipMemcpy(d_stoich_rowptrs, stoich.row_ptrs, (stoich.n_rows + 1) * sizeof(int), hipMemcpyHostToDevice);

        std::cout << "Copied stoich to device.\n";

        // Temporary workspace for matrix generation
        int *iwsp;
        hipMalloc(&iwsp, n_states * sizeof(int));
        CUDACHKERR();

        // Get the max number of threads that can fit to a block
        int max_block_size, num_blocks;

        int device_id;

        hipGetDevice(&device_id);
        CUDACHKERR();
        hipDeviceGetAttribute(&max_block_size, hipDeviceAttributeMaxThreadsPerBlock, device_id);
        CUDACHKERR();
        std::cout << "Max block size = " << max_block_size << "\n";

        num_blocks = (size_t) std::ceil(n_states / (max_block_size * 1.0));

        // Initialize dimensions
        nst = n_states;
        nr = n_reactions;
        ns = n_species;

        std::cout << "nst = " << nst << "nr = " << nr << "ns = " << ns << "\n";

        term.resize(n_reactions);

        // Generate the state space
        fsp_get_states << < num_blocks, max_block_size, n_species * sizeof(size_t) >> >
                                                        (states, n_species, n_states, fsp_dim);
        CUDACHKERR();
        hipDeviceSynchronize();

        std::cout << "State space generation successful.\n";

        for (size_t ir{0}; ir < nr; ++ir) {
            // Initialize CSR data structure for the ir-th matrix
            hipMallocManaged((void **) &((term.at(ir)).row_ptrs), (n_states + 1) * sizeof(int));
            CUDACHKERR();
        }

        for (int ir{0}; ir < nr; ++ir) {
            term.at(ir).n_cols = n_states;
            term.at(ir).n_rows = n_states;

            // Count nonzero entries and store off-diagonal col indices to the temporary workspace
            size_t shared_size = ns * sizeof(size_t);
            fspmat_component_get_nnz_per_row << < num_blocks, max_block_size, shared_size >> > (term[ir].row_ptrs +
                                                                                                1, iwsp, states, ir, n_states, n_species, fsp_dim,
                    d_stoich_vals, d_stoich_colidxs, d_stoich_rowptrs);
            CUDACHKERR();
            hipDeviceSynchronize();
            CUDACHKERR();

            std::cout << "ir = " << ir << " get_nnz_per_row finished.\n";
            // Use cumulative sum to determine the values of the row pointers in CSR

            term[ir].row_ptrs[0] = 0;
            thrust::inclusive_scan(term[ir].row_ptrs, term[ir].row_ptrs + (nst + 1), term[ir].row_ptrs);
            CUDACHKERR();
            std::cout << "ir = " << ir << " inclusive scan finished.\n";

            // Fill the column indices and values
            int nnz;

            hipMemcpy(&nnz, term[ir].row_ptrs + nst, sizeof(int), hipMemcpyDeviceToHost); CUDACHKERR();

            std::cout << "nnz = " << nnz << "\n";

            hipMallocManaged(&(term[ir].vals), nnz * sizeof(double));
            CUDACHKERR();
            hipMallocManaged(&(term[ir].col_idxs), nnz * sizeof(int));
            CUDACHKERR();

            fspmat_component_fill_data_csr << < num_blocks, max_block_size >> >
                                                            (term[ir].vals, term[ir].col_idxs, term[ir].row_ptrs, nst, ir, iwsp, states, ns,
                                                                    prop_func);
            CUDACHKERR();

            hipDeviceSynchronize();
            CUDACHKERR();
            std::cout << "ir = " << ir << " fill_data_csr finished.\n";
        }

        hipFree(d_stoich_colidxs);
        CUDACHKERR();
        hipFree(d_stoich_rowptrs);
        CUDACHKERR();
        hipFree(d_stoich_vals);
        CUDACHKERR();
        hipFree(iwsp);
        CUDACHKERR();
    }

    void FSPMat::action(double t, double *x, double *y) {

    }

    // Destructor
    FSPMat::~FSPMat() {
        for (size_t i{0}; i < nr; ++i) {
            hipFree(term[i].col_idxs);
            CUDACHKERR();
            hipFree(term[i].row_ptrs);
            CUDACHKERR();
            hipFree(term[i].vals);
            CUDACHKERR();
        }
        hipsparseDestroyMatDescr(cusparse_descr);
        CUDACHKERR();
    }

    __global__

    void fsp_get_states(int *d_states, size_t dim, size_t n_states, size_t *n_bounds) {

        extern __shared__
        size_t n_bounds_copy[];

        size_t ti = threadIdx.x;
        size_t indx = blockIdx.x * blockDim.x + ti;

        if (ti < dim) {
            n_bounds_copy[ti] = n_bounds[ti];
        }

        __syncthreads();

        if (indx < n_states) {
            indx2state(indx, &d_states[indx * dim], dim, &n_bounds[0]);
        }
    }

    __host__
    __device__

    void reachable_state(int *state, int *rstate, int reaction, int direction,
                         int n_species, int *stoich_val, int *stoich_colidxs, int *stoich_rowptrs) {
        for (int k{0}; k < n_species; ++k) {
            rstate[k] = state[k];
        }
        for (int i = stoich_rowptrs[reaction]; i < stoich_rowptrs[reaction + 1]; ++i) {
            rstate[stoich_colidxs[i]] += direction * stoich_val[i];
        }
    }

    __global__

    void
    fspmat_component_get_nnz_per_row(int *nnz_per_row, int *off_indx, int *states, int reaction, size_t n_rows,
                                     size_t n_species, size_t *fsp_bounds,
                                     int *stoich_vals, int *stoich_colidxs, int *stoich_rowptrs) {
        extern __shared__
        size_t wsp[];

        size_t tix = threadIdx.x;
        size_t tid = blockDim.x * blockIdx.x + tix;

        size_t *fsp_bounds_copy = &wsp[0];

        if (tix < n_species)
        {
            fsp_bounds_copy[tix] = fsp_bounds[tix];
        }


        __syncthreads();


        int *state;


        if (tid < n_rows) {

            state = &states[tid * n_species];

            indx2state(tid, &state[0], n_species, fsp_bounds_copy);

            reachable_state(state, state, reaction, -1,
                            n_species, stoich_vals, stoich_colidxs, stoich_rowptrs);

            bool reachable = true;
            for (size_t k{0}; k < n_species; ++k) {
                reachable = reachable && ((state[k] >= 0) || (state[k] <= fsp_bounds_copy[k]));
            }

            nnz_per_row[tid] = 1;
            if (reachable) {
                off_indx[tid] = state2indx(state, n_species, fsp_bounds_copy);
                nnz_per_row[tid] += 1;
            } else {
                off_indx[tid] = -1;
            }

            reachable_state(state, state, reaction, 1,
                            n_species, stoich_vals, stoich_colidxs, stoich_rowptrs);

//            printf("tid = %d    nnz[tid] = %d \n", (int) tid, nnz_per_row[tid]);
        }
    }

    __global__

    void
    fspmat_component_fill_data_csr(double *values, int *col_indices, int *row_ptrs, size_t n_rows, int reaction,
                                   int *off_diag_indices, int *states, size_t dim, PropFun propensity) {

        size_t tid = blockDim.x * blockIdx.x + threadIdx.x;

        int off_diag_indx, rowptr, i_diag, i_offdiag;
        int *state;

        if (tid < n_rows) {
            off_diag_indx = off_diag_indices[tid];
            rowptr = row_ptrs[tid];

            if (off_diag_indx >= 0) {

                if (off_diag_indx > tid) {
                    i_diag = rowptr;
                    i_offdiag = rowptr + 1;
                } else {
                    i_diag = rowptr + 1;
                    i_offdiag = rowptr;
                }

                state = states + dim * tid;
                values[i_diag] = propensity(state, reaction);
                values[i_diag] *= -1.0;

                state = states + dim * off_diag_indx;
                values[i_offdiag] = propensity(state, reaction);

                col_indices[i_diag] = (int) tid;
                col_indices[i_offdiag] = off_diag_indx;
            } else {
                state = states + dim * tid;
                values[rowptr] = propensity(state, reaction);
                values[rowptr] *= -1.0;
                col_indices[rowptr] = (int) tid;
            }
        }
        __syncthreads();
    }
}