#include "hip/hip_runtime.h"
#include "KryExpvFSP.h"


namespace cuFSP {

    KryExpvFSP::KryExpvFSP(double _t_final, MVFun &_matvec, thrust_dvec &_v, int _m, double _tol,
                           bool _iop, int _q_iop, double _anorm) :
            t_final(_t_final),
            matvec(_matvec),
            sol_vec(_v),
            m(_m),
            tol(_tol),
            IOP(_iop),
            q_iop(_q_iop),
            anorm(_anorm) {
        n = (int) _v.size();
        hipMalloc(&wsp, (n * (m + 2) + (m + 2) * (m + 2)) * sizeof(double));

        // Pointers to the Krylov vectors
        V.resize((size_t) m + 2);
        V[0] = wsp;
        for (size_t i{1}; i < m + 2; ++i) {
            V[i] = V[i - 1] + n;
        }

        hipHostMalloc((void **) &pinned_H, (m + 2) * (m + 2) * sizeof(double));
        hipHostMalloc((void **) &pinned_F, (m + 2) * (m + 2) * sizeof(double));

        // Arma wrapper for Hessenberg matrix on host
        H = arma::Mat<double>(pinned_H, (size_t) m + 2, (size_t) m + 2, false, true);
        F = arma::Mat<double>(pinned_F, (size_t) m + 2, (size_t) m + 2, false, true);

        hipblasStatus_t stat = hipblasCreate(&cublas_handle);
        CUBLASCHKERR(stat);
        CUDACHKERR();
    }

    void KryExpvFSP::step() {
        hipblasStatus_t stat;

        stat = hipblasDnrm2(cublas_handle, n, (double *) thrust::raw_pointer_cast(&sol_vec[0]), 1, &beta);
        CUDACHKERR();
        CUBLASCHKERR(stat);

        if (i_step == 0) {
            xm = 1.0 / double(m);
            //double anorm { norm(A, 1) };
            double fact = pow((m + 1) / exp(1.0), m + 1) * sqrt(2 * (3.1416) * (m + 1));
            t_new = (1.0 / anorm) * pow((fact * tol) / (4.0 * beta * anorm), xm);
            btol = anorm * tol; // tolerance for happy breakdown
        }

        mb = m;
        double tau = std::min(t_final - t_now, t_new);

        stat = hipblasDscal(cublas_handle, (n * (m + 2) + (m + 2) * (m + 2)) , &zero, wsp, 1);
        CUBLASCHKERR(stat);

        double betainv = 1.0 / beta;
        stat = hipblasDcopy(cublas_handle, n, (double *) thrust::raw_pointer_cast(&sol_vec[0]), 1, V[0], 1);
        CUBLASCHKERR(stat);
        CUDACHKERR();
        stat = hipblasDscal(cublas_handle, n, &betainv, V[0], 1);
        CUBLASCHKERR(stat);
        CUDACHKERR();

        int istart = 0;
        double *d_H = V[m + 1] + n;
        /* Arnoldi loop */
        for (int j{0}; j < m; j++) {
            matvec(V[j], V[j + 1]);

            /* Orthogonalization */
            if (IOP) istart = (j >= q_iop - 1) ? j - q_iop + 1 : 0;

            stat = hipblasDgemv(cublas_handle, HIPBLAS_OP_T, n, (j - istart + 1), &one, V[istart], n, V[j + 1],
                                  1, &one, &d_H[istart + j * (m + 2)], 1);
            CUBLASCHKERR(stat);
            stat = hipblasDgemv(cublas_handle, HIPBLAS_OP_N, n, (j - istart + 1), &minus_one, V[istart], n,
                                  &d_H[istart + j * (m + 2)],
                                  1, &one, V[j + 1], 1);
            CUBLASCHKERR(stat);
            CUDACHKERR();

            //            s = norm(V[j + 1], 2);
            stat = hipblasDnrm2(cublas_handle, n, V[j + 1], 1, &s);
            CUBLASCHKERR(stat);
            CUDACHKERR();

            if (s < btol) {
                k1 = 0;
                mb = j + 1;
                tau = t_final - t_now;
#ifdef KEXPV_VERBOSE
                std::cout << "happy breakdown.\n";
#endif
                break;
            }

            hipMemcpy(&d_H[(j + 1) + j * (m + 2)], &s, sizeof(double), hipMemcpyHostToDevice);
            CUDACHKERR();
//            H(j + 1, j) = s;
            double sinv = 1.0 / s;
//            V[j + 1] = V[j + 1] / s;
            stat = hipblasDscal(cublas_handle, n, &sinv, V[j + 1], 1);

            CUBLASCHKERR(stat);
            CUDACHKERR();
        }

        hipMemcpy(H.colptr(0), d_H, (m + 2) * (m + 2) * sizeof(double), hipMemcpyDeviceToHost);
        CUDACHKERR();

        if (k1 != 0) {
            H((size_t) m + 1, (size_t) m) = 1.0;
            matvec(V[mb], V[mb + 1]);
            stat = hipblasDnrm2(cublas_handle, n, V[mb + 1], 1, &avnorm);
            CUBLASCHKERR(stat);
            CUDACHKERR();
        }

        size_t ireject{0};
        while (ireject < max_reject) {
            mx = mb + k1;
            arma::expmat(F, tau * H);
            if (k1 == 0) {
                err_loc = btol;
                break;
            } else {
                double phi1 = std::abs(beta * F((size_t) mx - 2, 0));
                double phi2 = std::abs(beta * F((size_t) mx - 1, 0) * avnorm);

                if (phi1 > phi2 * 10.0) {
                    err_loc = phi2;
                    xm = 1.0 / double(mx);
                } else if (phi1 > phi2) {
                    err_loc = (phi1 * phi2) / (phi1 - phi2);
                    xm = 1.0 / double(mx);
                } else {
                    err_loc = phi1;
                    xm = 1.0 / double(mx - 1);
                }
            }

            if (err_loc <= delta * tau * tol) {
                break;
            } else {
                tau = gamma * tau * pow(tau * tol / err_loc, xm);
                double s = pow(10.0, floor(log10(tau)) - 1);
                tau = ceil(tau / s) * s;
                if (ireject == max_reject) {
                    std::cout << "Maximum number of failed steps reached.";
                    t_now = t_final;
                    break;
                }
                ireject++;
            }
        }

        mx = mb + std::max(0, k1 - 1);
        double *F0 = V[m + 1] + n;
        hipMemcpy(F0, F.colptr(0), mx * sizeof(double), hipMemcpyHostToDevice);
        CUDACHKERR();

        stat = hipblasDgemv(cublas_handle, HIPBLAS_OP_N, n, mx, &beta, V[0], n, F0, 1, &zero,
                              (double *) thrust::raw_pointer_cast(&sol_vec[0]), 1);
        CUBLASCHKERR(stat);
        CUDACHKERR();


        t_now = t_now + tau;
        t_new = gamma * tau * pow(tau * tol / err_loc, xm);
        s = pow(10.0, floor(log10(t_new)) - 1.0);
        t_new = ceil(t_new / s) * s;

#ifdef KEXPV_VERBOSE
        //        std::cout << "t_now = " << t_now << " err_loc = " << err_loc << "\n";
                printf("i_step = %d \n t_now = %.2f err_loc = %.2e \n", i_step , t_now, err_loc);
#endif
        i_step++;

    }

    void KryExpvFSP::solve() {
        while (!final_time_reached()) {
            step();
        }
    }

    KryExpvFSP::~KryExpvFSP() {
        hipFree(wsp);
        CUDACHKERR();
        if (pinned_H) {
            hipHostFree(pinned_H);
            CUDACHKERR();
        }
        if (pinned_F) {
            hipHostFree(pinned_F);
            CUDACHKERR();
        }
        hipblasDestroy(cublas_handle);
    }
}
