#include "hip/hip_runtime.h"
//
// Created by Huy Vo on 10/30/18.
//
#include <armadillo>
#include <iostream>
#include <iomanip>
#include <time.h>
#include <hip/hip_runtime.h>
#include "FSPMat.h"
#include "cme_util.h"
#include "FSPMat.h"
#include "hipsparse.h"
#include "thrust/transform.h"
#include "thrust/execution_policy.h"
#include "thrust/device_vector.h"
#include "cme_util.h"
#include "FSPMat.h"
#include "KryExpvFSP.h"

/* Parameters for the propensity functions */
const double ayx{2.6e-3}, axy{6.1e-3}, nyx{3e0}, nxy{2.1e0},
        kx0{2.2e-3}, kx{1.7e-2}, dx{3.8e-4}, ky0{6.8e-5}, ky{1.6e-2}, dy{3.8e-4};

__device__ __host__
double toggle_propensity(int *x, int reaction) {
    double prop_val;
    switch (reaction) {
        case 0:
            prop_val = 1.0;
            break;
        case 1:
            prop_val = 1.0 / (1.0 + ayx*std::pow(1.0 * x[1], nyx));
            break;
        case 2:
            prop_val = 1.0 * x[0];
            break;
        case 3:
            prop_val = 1.0;
            break;
        case 4:
            prop_val = 1.0 / (1.0 + axy*std::pow(1.0 * x[0], nxy));
            break;
        case 5:
            prop_val = 1.0 * x[1];
            break;
    }
    return prop_val;
}

__device__ cuFSP::PropFun prop_pointer = &toggle_propensity;

__device__ __host__
void t_func(double t, double* out){
//    return {(1.0 + std::cos(t))*kx0, kx, dx, (1.0 + std::sin(t))*ky0, ky, dy};
    out[0] = kx0;
    out[1] = kx;
    out[2] = dx;
    out[3] = ky0;
    out[4] = ky;
    out[5] = dy;
}

int main()
{
    int n_species = 2;
    int n_reactions = 6;

    int stoich_vals[] = {1, 1, -1, 1,1, -1};
    int stoich_colidxs[] = {0, 0, 0, 1, 1, 1};
    int stoich_rowptrs[] = {0, 1, 2, 3, 4, 5,6};

    cuFSP::CSRMatInt stoich;
    stoich.vals = &stoich_vals[0];
    stoich.col_idxs = &stoich_colidxs[0];
    stoich.row_ptrs = &stoich_rowptrs[0];
    stoich.n_rows = 6;
    stoich.n_cols = 2;
    stoich.nnz = 6;

    int n_bounds[] = {1<<8, 1<<8};

    std::cout << n_bounds[0] << " " << n_bounds[1] << "\n";
    int n_states = cuFSP::rect_fsp_num_states(n_species, n_bounds);
    std::cout << "Total number of states:" << n_states << "\n";

    cuFSP::PropFun host_prop_ptr;
    hipMemcpyFromSymbol(&host_prop_ptr, HIP_SYMBOL(prop_pointer), sizeof(cuFSP::PropFun)); CUDACHKERR();
    cuFSP::FSPMat A
    (n_reactions, n_species, n_bounds,
            stoich, &t_func, host_prop_ptr, cuFSP::HYB); CUDACHKERR();
    hipDeviceSynchronize();

    thrust::device_vector<double> v(n_states);
    thrust::fill(v.begin(), v.end(), 0.0); CUDACHKERR();
    v[0] = 1.0;
    hipDeviceSynchronize(); CUDACHKERR();

    double t_final = 8*3600;
    double tol = 1.0e-8;
    int m = 30;
    std::function<void (double*, double*)> matvec = [&] (double*x, double* y) {
        A.action(1.0, x, y);
        return;
    };

    cuFSP::KryExpvFSP expv(t_final, matvec, v, m, tol, true);
    expv.solve();
    hipDeviceSynchronize();
    double vsum = thrust::reduce(v.begin(), v.end());
    assert(std::abs(1.0 - vsum) <= 1.0e-10);

    return 0;
}