#include "hip/hip_runtime.h"
//
// Created by Huy Vo on 10/30/18.
//
#include <armadillo>
#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>
#include "FSPMat.h"
#include "cme_util.h"
#include "FSPMat.h"
#include "hipsparse.h"
#include "../src/cme_util.h"
#include "../src/FSPMat.h"

__device__ __host__
double toggle_propensity(int *x, int reaction) {
    double prop_val;
    switch (reaction) {
        case 0:
            prop_val = 1.0 / (1.0 + std::pow(1.0 * x[1], 2.0));
            break;
        case 1:
            prop_val = 1.0 * x[0];
            break;
        case 2:
            prop_val = 1.0 / (1.0 + std::pow(1.0 * x[0], 2.0));
            break;
        case 3:
            prop_val = 1.0 * x[1];
            break;
    }
    return prop_val;
}

__device__ cuFSP::PropFun prop_pointer = &toggle_propensity;

Col<double> t_func(double t){
    return arma::Col<double>({1.0, 1.0, 1.0, 1.0});
}

int main()
{
//    hipsparseHandle_t cusparse_handle;
//    hipStream_t stream;
//
//    // Initialize cuSparse handle and bind to stream
//    hipsparseCreate(&cusparse_handle); CUDACHKERR();
//    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking); CUDACHKERR();
//    hipsparseSetStream(cusparse_handle, stream); CUDACHKERR();

    hipDeviceSynchronize();

    size_t n_species = 2;
    size_t n_reactions = 4;

    int stoich_vals[] = {1, -1, 1, -1};
    int stoich_colidxs[] = {0, 0, 1, 1};
    int stoich_rowptrs[] = {0, 1, 2, 3, 4};

    cuFSP::cuda_csr_mat_int stoich;
    stoich.vals = &stoich_vals[0];
    stoich.col_idxs = &stoich_colidxs[0];
    stoich.row_ptrs = &stoich_rowptrs[0];
    stoich.n_rows = 4;
    stoich.n_cols = 2;

    size_t *n_bounds;
    int *states;

    hipMallocManaged(&n_bounds, n_species*sizeof(size_t));

    n_bounds[0] = (1 << 10) - 1;
    n_bounds[1] = (1 << 10) - 1;

    std::cout << n_bounds[0] << " " << n_bounds[1] << "\n";

    size_t n_states = 1;
    for (size_t i{0}; i < n_species; ++i) {
        n_states *= (n_bounds[i] + 1);
    }
    std::cout << "Total number of states:" << n_states << "\n";

    hipMallocManaged(&states, n_states * n_species * sizeof(int)); CUDACHKERR();

    cuFSP::PropFun host_prop_ptr;
    hipMemcpyFromSymbol(&host_prop_ptr, HIP_SYMBOL(prop_pointer), sizeof(cuFSP::PropFun)); CUDACHKERR();

    cuFSP::FSPMat A
//    (cusparse_handle, stream,
                    (states, n_states, n_reactions, n_species, n_bounds,
            stoich, t_func, host_prop_ptr);

    hipDeviceSynchronize();
    std::cout << "Matrix generation successful.\n";

//    hipsparseDestroy(cusparse_handle); CUDACHKERR();
//    hipStreamDestroy(stream); CUDACHKERR();
    hipFree(states); CUDACHKERR();
    hipFree(n_bounds); CUDACHKERR();
//    hipDeviceReset();
    return 0;
}