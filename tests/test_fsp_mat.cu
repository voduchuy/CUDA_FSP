#include "hip/hip_runtime.h"
//
// Created by Huy Vo on 10/30/18.
//
#include <armadillo>
#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>
#include "FSPMat.h"
#include "cme_util.h"
#include "FSPMat.h"
#include "hipsparse.h"
#include "hipblas.h"
#include "../src/cme_util.h"
#include "../src/FSPMat.h"
#include "thrust/transform.h"
#include "thrust/execution_policy.h"
#include "thrust/device_vector.h"

/* Parameters for the propensity functions */
const double ayx{2.6e-3}, axy{6.1e-3}, nyx{3e0}, nxy{2.1e0},
        kx0{2.2e-3}, kx{1.7e-2}, dx{3.8e-4}, ky0{6.8e-5}, ky{1.6e-2}, dy{3.8e-4};

__device__ __host__
double toggle_propensity(int *x, int reaction) {
    double prop_val;
    switch (reaction) {
        case 0:
            prop_val = 1.0;
            break;
        case 1:
            prop_val = 1.0 / (1.0 + ayx*pow(1.0 * x[1], nyx));
            break;
        case 2:
            prop_val = 1.0 * x[0];
            break;
        case 3:
            prop_val = 1.0;
            break;
        case 4:
            prop_val = 1.0 / (1.0 + axy*pow(1.0 * x[0], nxy));
            break;
        case 5:
            prop_val = 1.0 * x[1];
            break;
        default:
            prop_val = 0.0;
            break;
    }
    return prop_val;
}

__device__ __host__
double toggle_propensity_factor(int x, int species, int reaction) {
    double prop_val = 1.0;
    switch (reaction) {
        case 0:
            break;
        case 1:
            if (species == 1) {
                prop_val = 1.0 / (1.0 + ayx*pow(1.0 * x, nyx));
            }
            break;
        case 2:
            if (species == 0){
                prop_val = 1.0*x;
            }
            break;
        case 3:
            break;
        case 4:
            if (species == 0) {
                prop_val = 1.0 / (1.0 + axy*pow(1.0 * x, nxy));
            }
            break;
        case 5:
            if (species == 1){
                prop_val = 1.0*x;
            }
            break;
        default:
            break;
    }
    return prop_val;
}

__device__ cuFSP::PropFun prop_pointer = &toggle_propensity;

__device__ __host__
void t_func(double t, double* out){
//    return {(1.0 + std::cos(t))*kx0, kx, dx, (1.0 + std::sin(t))*ky0, ky, dy};
    out[0] = kx0;
    out[1] = kx;
    out[2] = dx;
    out[3] = ky0;
    out[4] = ky;
    out[5] = dy;
}

int main()
{
    int n_species = 2;
    int n_reactions = 6;

    int stoich_vals[] = {1, 1, -1, 1, 1, -1};
    int stoich_colidxs[] = {0, 0, 0, 1, 1, 1};
    int stoich_rowptrs[] = {0, 1, 2, 3, 4, 5, 6};

    cuFSP::CSRMatInt stoich;
    stoich.vals = &stoich_vals[0];
    stoich.col_idxs = &stoich_colidxs[0];
    stoich.row_ptrs = &stoich_rowptrs[0];
    stoich.n_rows = 6;
    stoich.n_cols = 2;
    stoich.nnz = 6;

    int *states;

    int* n_bounds = new int[2];
    n_bounds[0] = (1<<12) - 1;
    n_bounds[1] = (1<<11) - 1;
    std::cout << n_bounds[0] << " " << n_bounds[1] << "\n";

    int n_states = cuFSP::rect_fsp_num_states(n_species, n_bounds);
    std::cout << "Total number of states:" << n_states << "\n";
    hipMalloc(&states, n_states * n_species * sizeof(int)); CUDACHKERR();

    cuFSP::PropFun host_prop_ptr;
    hipMemcpyFromSymbol(&host_prop_ptr, HIP_SYMBOL(prop_pointer), sizeof(cuFSP::PropFun)); CUDACHKERR();

    cuFSP::FSPMat A
    (states, n_states, n_reactions, n_species, n_bounds,
            stoich, &t_func, host_prop_ptr);
    hipDeviceSynchronize(); CUDACHKERR();
    std::cout << "CUDA_CSR matrix generation successful.\n";

    cuFSP::FSPMat A2
            (states, n_states, n_reactions, n_species, n_bounds,
             stoich, &t_func, host_prop_ptr, cuFSP::HYB);
    hipDeviceSynchronize(); CUDACHKERR();
    std::cout << "HYB matrix generation successful.\n";

    cuFSP::FSPMat A3
            (states, n_states, n_reactions, n_species, n_bounds, stoich, &t_func, &toggle_propensity_factor, cuFSP::KRONECKER);
    hipDeviceSynchronize(); CUDACHKERR();
    std::cout << "KRON matrix generation successful.\n";

    thrust::device_vector<double> v(n_states);
    thrust::device_vector<double> w(n_states);
    thrust::device_vector<double> w2(n_states);
    thrust::device_vector<double> w3(n_states);
    thrust::fill(v.begin(), v.end(), 0.0); CUDACHKERR();
    thrust::fill(w.begin(), w.end(), 0.0); CUDACHKERR();
    thrust::fill(w2.begin(), w2.end(), 0.0); CUDACHKERR();
    thrust::fill(w3.begin(), w3.end(), 0.0); CUDACHKERR();
    hipDeviceSynchronize(); CUDACHKERR();

    A.action(1.0, (double*) thrust::raw_pointer_cast(&v[0]), (double*) thrust::raw_pointer_cast(&w[0])); CUDACHKERR();
    double sum = thrust::reduce(w.begin(), w.end());
    hipDeviceSynchronize();
    std::cout << "sum = " << sum << "\n";
    assert( std::abs(sum) <= 1.0e-14);

    A2.action(1.0, (double*) thrust::raw_pointer_cast(&v[0]), (double*) thrust::raw_pointer_cast(&w2[0])); CUDACHKERR();
    sum = thrust::reduce(w2.begin(), w2.end());
    hipDeviceSynchronize();
    std::cout << "sum = " << sum << "\n";
    assert( std::abs(sum) <= 1.0e-14);

    A3.action(1.0, (double*) thrust::raw_pointer_cast(&v[0]), (double*) thrust::raw_pointer_cast(&w3[0])); CUDACHKERR();
    sum = thrust::reduce(w3.begin(), w3.end());
    hipDeviceSynchronize();
    std::cout << "sum = " << sum << "\n";
    assert( std::abs(sum) <= 1.0e-14);

    thrust::fill(v.begin(), v.end(), 1.0); CUDACHKERR();

    A.action(1.0, (double*) thrust::raw_pointer_cast(&v[0]), (double*) thrust::raw_pointer_cast(&w[0])); CUDACHKERR();
    sum = thrust::reduce(w.begin(), w.end());
    hipDeviceSynchronize();
    std::cout << "sum = " << sum << "\n";

    A2.action(1.0, (double*) thrust::raw_pointer_cast(&v[0]), (double*) thrust::raw_pointer_cast(&w2[0])); CUDACHKERR();
    sum = thrust::reduce(w2.begin(), w2.end());
    hipDeviceSynchronize();
    std::cout << "sum = " << sum << "\n";

    A3.action(1.0, (double*) thrust::raw_pointer_cast(&v[0]), (double*) thrust::raw_pointer_cast(&w3[0])); CUDACHKERR();
    sum = thrust::reduce(w3.begin(), w3.end());
    hipDeviceSynchronize();
    std::cout << "sum = " << sum << "\n";

    hipblasDaxpy(n_states, -1.0, (double*) thrust::raw_pointer_cast(&w2[0]), 1, (double*) thrust::raw_pointer_cast(&w[0]), 1);
    CUDACHKERR();

    double error_l2;
    error_l2 = hipblasDnrm2(n_states, (double*) thrust::raw_pointer_cast(&w[0]), 1); CUDACHKERR();
    std::cout << "error_l2 = " << error_l2 << "\n";
    assert(error_l2 <= 1.0e-14);

    hipblasDaxpy(n_states, -1.0, (double*) thrust::raw_pointer_cast(&w2[0]), 1, (double*) thrust::raw_pointer_cast(&w3[0]), 1);
    CUDACHKERR();

    error_l2 = hipblasDnrm2(n_states, (double*) thrust::raw_pointer_cast(&w3[0]), 1); CUDACHKERR();
    std::cout << "error_l2 = " << error_l2 << "\n";
    assert(error_l2 <= 1.0e-12);

    std::cout << "Matvec test successful.\n";

    hipFree(states); CUDACHKERR();
    delete[] n_bounds;
    return 0;
}